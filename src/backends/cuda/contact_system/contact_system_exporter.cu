#include <contact_system/contact_system_exporter.h>
#include <contact_system/contact_system_feature.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(ContactSystemExporter);

void ContactSystemExporter::do_build()
{
    m_global_trajectory_filter = require<GlobalTrajectoryFilter>();
    m_global_contact_manager   = require<GlobalContactManager>();


    auto overrider = std::make_shared<ContactSystemFeatureOverrider>(this);
    auto feature   = std::make_shared<core::ContactSystemFeature>(overrider);
    features().insert(feature);

    on_init_scene(
        [&]
        {
            m_simplex_trajectory_filter =
                m_global_trajectory_filter->find<SimplexTrajectoryFilter>();

            m_vertex_half_plane_trajectory_filter =
                m_global_trajectory_filter->find<VertexHalfPlaneTrajectoryFilter>();
        });
}

void ContactSystemExporter::get_contact_gradient(geometry::Geometry& vert_grad)
{
    auto& g = m_global_contact_manager->m_impl.sorted_contact_gradient;

    vert_grad.instances().resize(g.doublet_count());
    auto i = vert_grad.instances().find<IndexT>("i");
    if(!i)
    {
        i = vert_grad.instances().create<IndexT>("i");
    }
    auto grad = vert_grad.instances().find<Vector3>("grad");
    if(!grad)
    {
        grad = vert_grad.instances().create<Vector3>("grad");
    }

    auto i_view = view(*i);
    g.indices().copy_to(i_view.data());

    auto grad_view = view(*grad);
    g.values().copy_to(grad_view.data());
}

void ContactSystemExporter::get_contact_hessian(geometry::Geometry& vert_hess)
{
    auto& h = m_global_contact_manager->m_impl.sorted_contact_hessian;

    vert_hess.instances().resize(h.triplet_count());
    auto i = vert_hess.instances().find<IndexT>("i");
    if(!i)
    {
        i = vert_hess.instances().create<IndexT>("i");
    }
    auto j = vert_hess.instances().find<IndexT>("j");
    if(!j)
    {
        j = vert_hess.instances().create<IndexT>("j");
    }
    auto hess = vert_hess.instances().find<Matrix3x3>("hess");
    if(!hess)
    {
        hess = vert_hess.instances().create<Matrix3x3>("hess");
    }
    auto i_view    = view(*i);
    auto j_view    = view(*j);
    auto hess_view = view(*hess);

    h.row_indices().copy_to(i_view.data());
    h.col_indices().copy_to(j_view.data());
    h.values().copy_to(hess_view.data());
}

void ContactSystemExporter::get_contact_primtives(std::string_view    prim_type,
                                                  geometry::Geometry& prims)
{
    auto supported_types = get_contact_primitive_types();
    if(std::find(supported_types.begin(), supported_types.end(), prim_type)
       == supported_types.end())
    {
        spdlog::warn("Unsupported contact primitive type: {}. Supported types are: [{}], ignore output.",
                     prim_type,
                     fmt::join(supported_types, ", "));
        return;
    }
    // add type to geometry.meta()
    auto type = prims.meta().find<std::string>("type");
    if(!type)
    {
        type = prims.meta().create<std::string>("type");
    }
    view(*type)[0] = prim_type;

    if(m_simplex_trajectory_filter)
    {
        if(prim_type == "PP")
        {
            auto PPs = m_simplex_trajectory_filter->PPs();

            prims.instances().resize(PPs.size());
            auto topo = prims.instances().find<Vector2i>("topo");
            if(!topo)
            {
                topo = prims.instances().create<Vector2i>("topo");
            }
            auto topo_view = view(*topo);
            PPs.copy_to(topo_view.data());
        }
        else if(prim_type == "PE")
        {
            auto PEs = m_simplex_trajectory_filter->PEs();

            prims.instances().resize(PEs.size());
            auto topo = prims.instances().find<Vector3i>("topo");
            if(!topo)
            {
                topo = prims.instances().create<Vector3i>("topo");
            }
            auto topo_view = view(*topo);
            PEs.copy_to(topo_view.data());
        }
        else if(prim_type == "PT")
        {
            auto PTs = m_simplex_trajectory_filter->PTs();

            prims.instances().resize(PTs.size());
            auto topo = prims.instances().find<Vector4i>("topo");
            if(!topo)
            {
                topo = prims.instances().create<Vector4i>("topo");
            }
            auto topo_view = view(*topo);
            PTs.copy_to(topo_view.data());
        }
        else if(prim_type == "EE")
        {
            auto EEs = m_simplex_trajectory_filter->EEs();

            prims.instances().resize(EEs.size());
            auto topo = prims.instances().find<Vector4i>("topo");
            if(!topo)
            {
                topo = prims.instances().create<Vector4i>("topo");
            }
            auto topo_view = view(*topo);
            EEs.copy_to(topo_view.data());
        }
    }

    if(m_vertex_half_plane_trajectory_filter)
    {
        if(prim_type == "PH")
        {
            auto PHs = m_vertex_half_plane_trajectory_filter->PHs();

            prims.instances().resize(PHs.size());
            auto topo = prims.instances().find<Vector2i>("topo");
            if(!topo)
            {
                topo = prims.instances().create<Vector2i>("topo");
            }
            auto topo_view = view(*topo);
            PHs.copy_to(topo_view.data());
        }
    }
}

vector<std::string> ContactSystemExporter::get_contact_primitive_types() const
{
    return {"PP", "PE", "PT", "EE", "PH"};
}
}  // namespace uipc::backend::cuda