#include <finite_element/finite_element_extra_constitution.h>
#include <uipc/builtin/attribute_name.h>
#include <finite_element/constitutions/discrete_shell_bending_function.h>
#include <numbers>
#include <utils/make_spd.h>
#include <utils/matrix_assembler.h>
#include <kernel_cout.h>

namespace std
{
// hash function for Vector2i
template <>
struct hash<uipc::Vector2i>
{
    size_t operator()(const uipc::Vector2i& v) const
    {
        size_t front = v[0];
        size_t end   = v[1];
        return front << 32 | end;
    }
};
}  // namespace std

namespace uipc::backend::cuda
{
class DiscreteShellBending final : public FiniteElementExtraConstitution
{
    static constexpr U64 DiscreteShellBendingUID = 17;
    using Base = FiniteElementExtraConstitution;

  public:
    using Base::Base;
    U64 get_uid() const noexcept override { return DiscreteShellBendingUID; }

    class InitInfo
    {
      public:
        bool        valid_bending() const { return oppo_verts.size() == 2; }
        IndexT      edge_index = -1;
        set<IndexT> oppo_verts;
        Float       stiffness = 0.0;
    };

    vector<Vector4i> h_stencils;  // X0, X1, X2, X3; (X1, X2) is middle edge
    vector<Float>    h_bending_stiffness;
    vector<Float>    h_rest_volumes;
    vector<Float>    h_rest_lengths;
    vector<Float>    h_h_bars;
    vector<Float>    h_theta_bars;
    vector<Float>    h_V_bars;

    muda::DeviceBuffer<Vector4i> stencils;  // X0, X1, X2, X3; (X1, X2) is middle edge
    muda::DeviceBuffer<Float> bending_stiffnesses;
    muda::DeviceBuffer<Float> rest_lengths;
    muda::DeviceBuffer<Float> h_bars;
    muda::DeviceBuffer<Float> theta_bars;
    muda::DeviceBuffer<Float> V_bars;

    virtual void do_build(BuildInfo& info) override {}

    virtual void do_init(FilteredInfo& info) override
    {
        namespace DSB = sym::discrete_shell_bending;

        using ForEachInfo = FiniteElementMethod::ForEachInfo;
        auto geo_slots    = world().scene().geometries();

        list<Vector4i> stencil_list;
        list<Float>    bending_stiffness_list;

        // 1) Retrieve Quad Stencils
        info.for_each(  //
            geo_slots,
            [&](const ForEachInfo& I, geometry::SimplicialComplex& sc)
            {
                unordered_map<Vector2i, InitInfo> stencil_map;  // Edge -> opposite vertices

                auto vertex_offset =
                    sc.meta().find<IndexT>(builtin::backend_fem_vertex_offset);
                UIPC_ASSERT(vertex_offset, "Vertex offset not found, why?");
                auto vertex_offset_v = vertex_offset->view().front();

                auto edges = sc.edges().topo().view();

                for(auto&& [i, e] : enumerate(edges))
                {
                    Vector2i E = e;
                    std::sort(E.begin(), E.end());

                    stencil_map[E].edge_index = i;
                }

                auto triangles = sc.triangles().topo().view();
                for(auto&& t : triangles)
                {
                    Vector3i T = t;
                    std::sort(T.begin(), T.end());

                    Vector2i E01 = {T[0], T[1]};
                    Vector2i E02 = {T[0], T[2]};
                    Vector2i E12 = {T[1], T[2]};

                    // insert opposite vertices
                    stencil_map[E01].oppo_verts.insert(T[2]);
                    stencil_map[E02].oppo_verts.insert(T[1]);
                    stencil_map[E12].oppo_verts.insert(T[0]);
                }

                auto bending_stiffnesses = sc.edges().find<Float>("bending_stiffness");
                UIPC_ASSERT(bending_stiffnesses, "Bending stiffness not found, why?");
                auto bs_view = bending_stiffnesses->view();

                for(auto&& [E, info] : stencil_map)
                {
                    if(info.valid_bending())
                    {
                        // X0, X1, X2, X3; (X1, X2) is middle edge
                        Vector4i stencil{*info.oppo_verts.begin(),    // X0
                                         E(0),                        // X1
                                         E(1),                        // X2
                                         *info.oppo_verts.rbegin()};  // X3

                        // convert to fem vertex index
                        stencil_list.push_back(stencil.array() + vertex_offset_v);

                        Float bs = bs_view[info.edge_index];
                        bending_stiffness_list.push_back(bs);
                    }
                }
            });

        // 2) Setup Invariant Data
        h_stencils.resize(stencil_list.size());
        h_bending_stiffness.resize(stencil_list.size());
        std::ranges::move(stencil_list, h_stencils.begin());
        std::ranges::move(bending_stiffness_list, h_bending_stiffness.begin());

        // 3) Setup Related Data
        span x_bars      = info.rest_positions();
        span thicknesses = info.thicknesses();
        h_rest_lengths.resize(h_stencils.size());
        h_h_bars.resize(h_stencils.size());
        h_theta_bars.resize(h_stencils.size());
        h_V_bars.resize(h_stencils.size());

        for(auto&& [i, stencil] : enumerate(h_stencils))
        {
            Vector3 X0         = x_bars[stencil[0]];
            Vector3 X1         = x_bars[stencil[1]];
            Vector3 X2         = x_bars[stencil[2]];
            Vector3 X3         = x_bars[stencil[3]];
            Float   thickness0 = thicknesses[stencil[0]];
            Float   thickness1 = thicknesses[stencil[1]];
            Float   thickness2 = thicknesses[stencil[2]];
            Float   thickness3 = thicknesses[stencil[3]];

            Float L0, V_bar, h_bar, theta_bar;
            DSB::compute_constants(L0,
                                   h_bar,
                                   theta_bar,
                                   V_bar,  //
                                   X0,
                                   X1,
                                   X2,
                                   X3,  //
                                   thickness0,
                                   thickness1,
                                   thickness2,
                                   thickness3);


            h_rest_lengths[i] = L0;
            h_h_bars[i]       = h_bar;
            h_theta_bars[i]   = theta_bar;
            h_V_bars[i]       = V_bar;
        }

        // 4) Copy to Device
        stencils.resize(h_stencils.size());
        stencils.view().copy_from(h_stencils.data());

        bending_stiffnesses.resize(h_bending_stiffness.size());
        bending_stiffnesses.view().copy_from(h_bending_stiffness.data());

        rest_lengths.resize(h_rest_lengths.size());
        rest_lengths.view().copy_from(h_rest_lengths.data());

        h_bars.resize(h_h_bars.size());
        h_bars.view().copy_from(h_h_bars.data());

        theta_bars.resize(h_theta_bars.size());
        theta_bars.view().copy_from(h_theta_bars.data());

        V_bars.resize(h_V_bars.size());
        V_bars.view().copy_from(h_V_bars.data());
    }

    virtual void do_report_extent(ReportExtentInfo& info) override
    {
        info.energy_count(stencils.size());  // Each quad has 1 energy
        info.stencil_dim(4);                 // Each quad has 4 vertices
    }

    virtual void do_compute_energy(ComputeEnergyInfo& info) override
    {
        using namespace muda;
        namespace DSB = sym::discrete_shell_bending;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.energies().size(),
                   [stencils = stencils.viewer().name("stencils"),
                    bending_stiffnesses = bending_stiffnesses.viewer().name("bending_stiffness"),
                    theta_bars = theta_bars.viewer().name("theta_bar"),
                    h_bars     = h_bars.viewer().name("h_bar"),
                    V_bars     = V_bars.viewer().name("V_bar"),
                    L0s        = rest_lengths.viewer().name("rest_lengths"),
                    xs         = info.xs().viewer().name("xs"),
                    energies   = info.energies().viewer().name("energies"),
                    dt         = info.dt()] __device__(int I)
                   {
                       Vector4i stencil   = stencils(I);
                       Float    kappa     = bending_stiffnesses(I);
                       Float    L0        = L0s(I);
                       Float    h_bar     = h_bars(I);
                       Float    theta_bar = theta_bars(I);
                       Float    V_bar     = V_bars(I);

                       Vector3 x0 = xs(stencil[0]);
                       Vector3 x1 = xs(stencil[1]);
                       Vector3 x2 = xs(stencil[2]);
                       Vector3 x3 = xs(stencil[3]);

                       Float E = DSB::E(x0, x1, x2, x3, L0, h_bar, theta_bar, kappa);
                       energies(I) = E * V_bar * dt * dt;
                   });
    }

    virtual void do_compute_gradient_hessian(ComputeGradientHessianInfo& info) override
    {
        using namespace muda;
        namespace DSB = sym::discrete_shell_bending;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(stencils.size(),
                   [stencils = stencils.viewer().name("stencils"),
                    bending_stiffnesses = bending_stiffnesses.viewer().name("bending_stiffness"),
                    theta_bars = theta_bars.viewer().name("theta_bar"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    h_bars = h_bars.viewer().name("h_bar"),
                    V_bars = V_bars.viewer().name("V_bar"),
                    L0s    = rest_lengths.viewer().name("rest_lengths"),
                    xs     = info.xs().viewer().name("xs"),
                    G3s    = info.gradients().viewer().name("gradients"),
                    H3x3s  = info.hessians().viewer().name("hessians"),
                    dt     = info.dt()] __device__(int I) mutable
                   {
                       Vector4i stencil   = stencils(I);
                       Float    kappa     = bending_stiffnesses(I);
                       Float    L0        = L0s(I);
                       Float    h_bar     = h_bars(I);
                       Float    theta_bar = theta_bars(I);
                       Float    V_bar     = V_bars(I);

                       Vector3 x0 = xs(stencil[0]);
                       Vector3 x1 = xs(stencil[1]);
                       Vector3 x2 = xs(stencil[2]);
                       Vector3 x3 = xs(stencil[3]);

                       Float Vdt2 = V_bar * dt * dt;

                       Vector12    G12;
                       Matrix12x12 H12x12;

                       DSB::dEdx(G12, x0, x1, x2, x3, L0, h_bar, theta_bar, kappa);
                       G12 *= Vdt2;
                       DoubletVectorAssembler DVA{G3s};
                       DVA.segment<4>(I * 4).write(stencil, G12);

                       DSB::ddEddx(H12x12, x0, x1, x2, x3, L0, h_bar, theta_bar, kappa);
                       H12x12 *= Vdt2;
                       make_spd(H12x12);

                       TripletMatrixAssembler TMA{H3x3s};
                       TMA.block<4, 4>(I * 4 * 4).write(stencil, H12x12);
                   });
    }
};


REGISTER_SIM_SYSTEM(DiscreteShellBending);
}  // namespace uipc::backend::cuda
