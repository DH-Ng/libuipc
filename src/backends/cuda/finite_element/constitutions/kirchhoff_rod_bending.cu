#include "hip/hip_runtime.h"
#include <finite_element/finite_element_extra_constitution.h>
#include <uipc/builtin/attribute_name.h>
#include <finite_element/constitutions/kirchhoff_rod_bending_function.h>
#include <numbers>
#include <utils/make_spd.h>
#include <utils/matrix_assembler.h>

#include <kernel_cout.h>
namespace uipc::backend::cuda
{
class KirchhoffRodBending final : public FiniteElementExtraConstitution
{
    static constexpr U64 KirchhoffRodBendingUID = 15;
    using Base = FiniteElementExtraConstitution;

  public:
    using Base::Base;
    U64 get_uid() const noexcept override { return KirchhoffRodBendingUID; }

    vector<Vector3i> h_hinges;
    vector<Float>    h_bending_stiffness;

    muda::DeviceBuffer<Vector3i> hinges;
    muda::DeviceBuffer<Float>    bending_stiffnesses;


    virtual void do_build(BuildInfo& info) override {}

    virtual void do_init(FilteredInfo& info) override
    {
        using ForEachInfo = FiniteElementMethod::ForEachInfo;
        auto geo_slots    = world().scene().geometries();


        list<Vector3i> hinge_list;  // X0, X1, X2
        list<Float>    bending_stiffness_list;

        info.for_each(  //
            geo_slots,
            [&](const ForEachInfo& I, geometry::SimplicialComplex& sc)
            {
                unordered_map<IndexT, set<IndexT>> hinge_map;  // Vertex -> Connected Vertices

                auto vertex_offset =
                    sc.meta().find<IndexT>(builtin::backend_fem_vertex_offset);
                UIPC_ASSERT(vertex_offset, "Vertex offset not found, why?");
                auto vertex_offset_v = vertex_offset->view().front();

                auto edges = sc.edges().topo().view();

                for(auto e : edges)
                {
                    auto v0 = e[0];
                    auto v1 = e[1];

                    hinge_map[v0].insert(v1);
                    hinge_map[v1].insert(v0);
                }

                auto bending_stiffnesses = sc.vertices().find<Float>("bending_stiffness");
                UIPC_ASSERT(bending_stiffnesses, "Bending stiffness not found, why?");

                auto bs_view = bending_stiffnesses->view();

                for(auto& [v, connected] : hinge_map)
                {
                    auto bs = bs_view[v];

                    if(connected.size() < 2)  // Not a hinge
                        continue;

                    for(auto v1 : connected)
                        for(auto v2 : connected)
                        {
                            if(v1 >= v2)  // Avoid duplicate
                                continue;

                            hinge_list.push_back({vertex_offset_v + v1,
                                                  vertex_offset_v + v,  // center vertex
                                                  vertex_offset_v + v2});
                            bending_stiffness_list.push_back(bs);
                        }
                }
            });

        // Setup data
        h_hinges.resize(hinge_list.size());
        h_bending_stiffness.resize(hinge_list.size());
        std::ranges::move(hinge_list, h_hinges.begin());
        std::ranges::move(bending_stiffness_list, h_bending_stiffness.begin());

        // Copy to device
        hinges.resize(h_hinges.size());
        hinges.view().copy_from(h_hinges.data());

        bending_stiffnesses.resize(h_bending_stiffness.size());
        bending_stiffnesses.view().copy_from(h_bending_stiffness.data());
    }

    virtual void do_report_extent(ReportExtentInfo& info) override
    {
        info.energy_count(hinges.size());  // Each hinge has 1 energy
        info.stencil_dim(3);               // Each hinge has 3 vertices
    }

    virtual void do_compute_energy(ComputeEnergyInfo& info) override
    {
        using namespace muda;
        namespace KRB = sym::kirchhoff_rod_bending;

        constexpr Float Pi = std::numbers::pi;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.energies().size(),
                   [hinges = hinges.viewer().name("hinges"),
                    bending_stiffnesses = bending_stiffnesses.viewer().name("bending_stiffness"),
                    thicknesses = info.thicknesses().viewer().name("thickness"),
                    xs          = info.xs().viewer().name("xs"),
                    x_bars      = info.x_bars().viewer().name("x_bars"),
                    energies    = info.energies().viewer().name("energies"),
                    dt          = info.dt(),
                    Pi] __device__(int I)
                   {
                       Vector3i hinge = hinges(I);
                       Float    k     = bending_stiffnesses(I) * dt * dt;
                       Float    r     = thicknesses(I);

                       Vector9 X;
                       X.segment<3>(0) = xs(hinge[0]);
                       X.segment<3>(3) = xs(hinge[1]);
                       X.segment<3>(6) = xs(hinge[2]);

                       Vector3 x0_bar = x_bars(hinge[0]);
                       Vector3 x1_bar = x_bars(hinge[1]);
                       Vector3 x2_bar = x_bars(hinge[2]);

                       // Rest length of the two edges
                       Float L0 = (x1_bar - x0_bar).norm() + (x2_bar - x1_bar).norm();

                       Float E;
                       KRB::E(E, k, X, L0, r, Pi);

                       energies(I) = E;
                   });
    }

    virtual void do_compute_gradient_hessian(ComputeGradientHessianInfo& info) override
    {
        using namespace muda;
        namespace KRB = sym::kirchhoff_rod_bending;

        constexpr Float Pi = std::numbers::pi;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(hinges.size(),
                   [hinges = hinges.viewer().name("hinges"),
                    bending_stiffnesses = bending_stiffnesses.viewer().name("bending_stiffness"),
                    thicknesses = info.thicknesses().viewer().name("thickness"),
                    xs          = info.xs().viewer().name("xs"),
                    x_bars      = info.x_bars().viewer().name("x_bars"),
                    G3s         = info.gradients().viewer().name("gradients"),
                    H3x3s       = info.hessians().viewer().name("hessians"),
                    dt          = info.dt(),
                    Pi] __device__(int I) mutable
                   {
                       Vector3i hinge = hinges(I);
                       Float    k     = bending_stiffnesses(I);
                       Float    r     = thicknesses(I);

                       Vector9 X;
                       X.segment<3>(0) = xs(hinge[0]);
                       X.segment<3>(3) = xs(hinge[1]);
                       X.segment<3>(6) = xs(hinge[2]);

                       Vector3 x0_bar = x_bars(hinge[0]);
                       Vector3 x1_bar = x_bars(hinge[1]);
                       Vector3 x2_bar = x_bars(hinge[2]);

                       // Rest length of the two edges
                       Float L0 = (x1_bar - x0_bar).norm() + (x2_bar - x1_bar).norm();

                       Float dt2 = dt * dt;

                       Vector9 G;
                       KRB::dEdX(G, k, X, L0, r, Pi);
                       G *= dt2;
                       DoubletVectorAssembler DVA{G3s};
                       DVA.segment<3>(I * 3).write(hinge, G);

                       Matrix9x9 H;
                       KRB::ddEddX(H, k, X, L0, r, Pi);

                       H *= dt2;
                       make_spd(H);
                       TripletMatrixAssembler TMA{H3x3s};
                       TMA.block<3, 3>(I * 3 * 3).write(hinge, H);
                   });
    }
};


REGISTER_SIM_SYSTEM(KirchhoffRodBending);
}  // namespace uipc::backend::cuda
